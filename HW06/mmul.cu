#include "mmul.h"

void mmul(hipblasHandle_t handle, const float* A, const float* B, float* C, int n){
    // alpha and beta are constant 1
    const float alpha = 1;
    const float beta = 1;

    // call cublas matrix multiplication function
    //	no transpose
    //	number of rows and columns of matrix A, B, and C are all n
    //	scalar 1 (alpha and beta)
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha, A, n, B, n, &beta, C, n);
    hipDeviceSynchronize(); // synchronize to make sure the cublas function is complete
}
