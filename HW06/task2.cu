#include "scan.cuh"
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <iostream>

int main(int argc, char* argv[]){
    // get parameters from commandline 
    unsigned int n = (unsigned int)atoi(argv[1]);
    unsigned int threads_per_block = (unsigned int)atoi(argv[2]);

    // input and output declaration
    float* input;
    float* output;

    // memory allocation of input and output array
    hipMallocManaged(&input, sizeof(float)*n);
    hipMallocManaged(&output, sizeof(float)*n);

    // random number generator
    srand(time(NULL));
    for(unsigned int i=0; i<n; i++){ // initialize input array with random numbers ranging from -1 to 1
	input[i] = ((float)rand() / RAND_MAX) * 2 - 1;
    }
    hipMemset(output, 0, sizeof(float)*n); // set output array 0

    // event creation for time measurement
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);

    hipEventRecord(start); // start recording time
    scan(input, output, n, threads_per_block); // call scan function
    hipEventRecord(end); // stop recording time
    hipEventSynchronize(end);

    float elapsed = 0; 
    hipEventElapsedTime(&elapsed, start, end); // get the elapsed time in milliseconds

    // print out the last element and elapsed time
    std::cout << output[n-1] << std::endl;
    std::cout << elapsed << std::endl;

    // free allocated memory
    hipFree(input);
    hipFree(output);

    return 0;
}




