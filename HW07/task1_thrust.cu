#include <iostream>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

int main(int argc, char* argv[]){
    int n = atoi(argv[1]); // read n from commandline

    thrust::host_vector<float> h_vec(n); // host vector of size n

    srand(time(NULL)); // random generator
    for(int i=0; i<n; i++) // initialize host vector with random numbers ranging from -1 to 1
	h_vec[i] = ((float)rand() / RAND_MAX) * 2 - 1;

    thrust::device_vector<float> d_vec = h_vec; // device vector copied from host vector 

    // event creation for timing measurement
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);

    hipEventRecord(start); // start recording time
    float ret = thrust::reduce(d_vec.begin(), d_vec.end(), 0.0, thrust::plus<float>()); // call reduce function
    hipEventRecord(end); // stop recording time
    hipEventSynchronize(end);

    float elapsed = 0;
    hipEventElapsedTime(&elapsed, start, end); // get elapsed time in milliseconds

    // destroy event for timing measurement
    hipEventDestroy(start);
    hipEventDestroy(end);

    std::cout << ret << std::endl; // print reduced result
    std::cout << elapsed << std::endl; // print elapsed time

    /*
    float _ret = 0; // manual reduce for test purpose
    for(int i=0; i<n; i++)
	_ret += h_vec[i];
    std::cout << _ret << std::endl;
    */
    return 0;
}


